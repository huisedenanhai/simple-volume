#include "hip/hip_runtime.h"
#include "random.h"
#include "render.h"
#include "vec_math.h"
#include <nanovdb/util/Ray.h>
#include <stdio.h>

__device__ float ray_march_transmittance(const nanovdb::FloatGrid *grid,
                                         const nanovdb::Ray<float> &wRay,
                                         float dt) {
  // transform the ray to the grid's index-space...
  nanovdb::Ray<float> iRay = wRay.worldToIndexF(*grid);
  // clip to bounds.
  if (iRay.clip(grid->tree().bbox()) == false) {
    return 1.0f;
  }
  // get an accessor.
  auto acc = grid->tree().getAccessor();
  // integrate along ray interval...
  float transmittance = 1.0f;
  for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
    float sigma = acc.getValue(nanovdb::Coord::Floor(iRay(t)));
    transmittance *= 1.0f - sigma * dt;
  }
  return transmittance;
}

__device__ __forceinline__ uint32_t reverse_bits_32(uint32_t n) {
  n = (n << 16) | (n >> 16);
  n = ((n & 0x00ff00ff) << 8) | ((n & 0xff00ff00) >> 8);
  n = ((n & 0x0f0f0f0f) << 4) | ((n & 0xf0f0f0f0) >> 4);
  n = ((n & 0x33333333) << 2) | ((n & 0xcccccccc) >> 2);
  n = ((n & 0x55555555) << 1) | ((n & 0xaaaaaaaa) >> 1);
  return n;
}

__device__ __forceinline__ float radical_inverse_base_2(uint32_t n) {
  return saturate(reverse_bits_32(n) * float(2.3283064365386963e-10));
}

// 0 <= i < n
__device__ __forceinline__ float2 hammersley_sample(uint32_t i, uint32_t n) {
  return make_float2((float)(i + 1) / (float)(n), radical_inverse_base_2(i));
}

__device__ __forceinline__ float
uniform_sample_cone(float u, float v, float cos_phi, float3 &d) {
  float theta = 2.0f * Pi * u;
  float y = 1.0f - u * (1.0f - cos_phi);
  auto r = sqrtf(max(0.0f, 1.0f - y * y));
  d.x = r * cosf(theta);
  d.y = y;
  d.z = r * sinf(theta);
  return 0.5f * InvPi / (1.0f - cos_phi);
}

// y goes upward
__device__ __forceinline__ float
uniform_sample_hemisphere(float u, float v, float3 &d) {
  return uniform_sample_cone(u, v, 0, d);
}

__device__ __forceinline__ float
uniform_sample_hemisphere(unsigned int &randState, float3 &d) {
  return uniform_sample_hemisphere(rnd(randState), rnd(randState), d);
}

__device__ __forceinline__ float
uniform_sample_sphere(float u, float v, float3 &d) {
  return uniform_sample_cone(u, v, -1.0f, d);
}

// y goes upward
__device__ __forceinline__ float
cosine_sample_hemisphere(float u, float v, float3 &d) {
  auto theta = 2.0f * Pi * u;
  auto r = sqrtf(v);
  auto y = sqrtf(max(1 - r * r, 0.0f));
  d.x = r * cosf(theta);
  d.y = y;
  d.z = r * sinf(theta);
  return y * InvPi;
}

__device__ __forceinline__ float
cosine_sample_hemisphere(unsigned int &randState, float3 &d) {
  return cosine_sample_hemisphere(rnd(randState), rnd(randState), d);
}

__device__ __forceinline__ float lerp(float a, float b, float t) {
  return a + t * (b - a);
}

__device__ __forceinline__ float4 lerp(float4 a, float4 b, float4 t) {
  return make_float4(lerp(a.x, b.x, t.x),
                     lerp(a.y, b.y, t.y),
                     lerp(a.z, b.z, t.z),
                     lerp(a.w, b.w, t.w));
}

__device__ __forceinline__ float4 lerp(float a, float b, float4 t) {
  return lerp(make_float4(a, a, a, a), make_float4(b, b, b, b), t);
}

__device__ __forceinline__ float inverse_lerp(float a, float b, float v) {
  return (v - a) / (b - a);
}

__device__ __forceinline__ float clamp(float v, float a, float b) {
  return max(min(v, b), a);
}

__device__ __forceinline__ float
sample_array(float *data, uint32_t count, float u) {
  uint32_t i = clamp(count * u, 0, count - 2);
  return lerp(data[i], data[i + 1], saturate(count * u - i));
}

__device__ __forceinline__ nanovdb::Ray<float> sample_camera_ray(
    const Scene &scene, int c, int r, float2 jitter = make_float2(0.0f, 0.0f)) {
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;

  float2 uv = make_float2((float(c) + jitter.x) / float(frame_width),
                          (float(r) + jitter.y) / float(frame_height));
  float aspect = float(frame_width) / float(frame_height);

  auto &camera_pos = scene.camera_pos;
  nanovdb::Vec3<float> origin(camera_pos[0], camera_pos[1], camera_pos[2]);
  nanovdb::Vec3<float> direction(uv.x - 0.5f, (uv.y - 0.5f) / aspect, -1.0);
  direction.normalize();
  nanovdb::Ray<float> wRay(origin, direction);
  return wRay;
}

__global__ void render_kernel_raymarching(Scene scene, float3 *image) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int r = blockIdx.y * blockDim.y + threadIdx.y;
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;
  const int index = r * frame_width + c;

  if ((c >= frame_width) || (r >= frame_height))
    return;

  nanovdb::Vec3<float> light_direction(
      scene.light_dir[0], scene.light_dir[1], scene.light_dir[2]);
  light_direction.normalize();
  auto grid = scene.volume_grid;

  nanovdb::Ray<float> wRay = sample_camera_ray(scene, c, r);
  // transform the ray to the grid's index-space...
  nanovdb::Ray<float> iRay = wRay.worldToIndexF(*grid);
  // clip to bounds.
  if (iRay.clip(grid->tree().bbox()) == false) {
    image[index] = make_float3(0, 0, 0);
    return;
  }
  // get an accessor.
  auto acc = grid->tree().getAccessor();
  // integrate along ray interval...
  float transmittance = 1.0f;
  float3 contrib = make_float3(0.0f, 0.0f, 0.0f);
  float dt = 0.5f;
  float3 color = make_float3(scene.phase_func.color[0],
                             scene.phase_func.color[1],
                             scene.phase_func.color[2]);

  for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
    auto iPos = iRay(t);
    float sigma = acc.getValue(nanovdb::Coord::Floor(iPos));
    contrib += ray_march_transmittance(
                   grid, {grid->indexToWorldF(iPos), light_direction}, dt) *
               transmittance * dt * sigma * color * 0.25f * InvPi;
    transmittance *= 1.0f - sigma * dt;
  }
  image[index] = contrib;
}

__device__ __forceinline__ float3 get_light_emission(const Scene &scene,
                                                     const float3 &direction) {
  float3 light_dir =
      make_float3(scene.light_dir[0], scene.light_dir[1], scene.light_dir[2]);
  float attenuation =
      dot(light_dir, direction) >= scene.light_cos_angle ? 1.0f : 0.0f;
  auto &light_color = scene.light_color;
  return attenuation *
         make_float3(light_color[0], light_color[1], light_color[2]);
}

__device__ __forceinline__ float sample_free_flight(float u, float mu) {
  return -logf(1.0f - u) / mu;
}

__device__ __forceinline__ float3
vec_as_float3(const nanovdb::Vec3<float> &vec) {
  return make_float3(vec[0], vec[1], vec[2]);
}

__device__ __forceinline__ nanovdb::Vec3<float>
float3_as_vec(const float3 &vec) {
  return nanovdb::Vec3<float>(vec.x, vec.y, vec.z);
}

__global__ void render_kernel_delta_tracking(Scene scene, float3 *image) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int r = blockIdx.y * blockDim.y + threadIdx.y;
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;
  const int index = r * frame_width + c;

  if ((c >= frame_width) || (r >= frame_height))
    return;

  unsigned int seed = tea<4>(index, 11424);
  int spp = scene.spp;
  float aspect = float(frame_width) / float(frame_height);
  auto grid = scene.volume_grid;
  float max_value = scene.max_value;
  float3 color = make_float3(scene.phase_func.color[0],
                             scene.phase_func.color[1],
                             scene.phase_func.color[2]);
  float3 contrib = make_float3(0.0f, 0.0f, 0.0f);
  auto accessor = grid->tree().getAccessor();
  for (int i = 0; i < spp; i++) {
    float2 jitter = hammersley_sample(i, spp);
    nanovdb::Ray<float> w_ray = sample_camera_ray(scene, c, r, jitter);
    float3 factor = make_float3(1.0f, 1.0f, 1.0f);

    bool miss = false;
    for (int bounce = 0; !miss && bounce < 30; bounce++) {
      nanovdb::Ray<float> i_ray = w_ray.worldToIndexF(*grid);
      if (i_ray.clip(grid->tree().bbox()) == false) {
        miss = true;
        break;
      }

      float t = i_ray.t0();

      while (t < i_ray.t1()) {
        t += sample_free_flight(rnd(seed), max_value);
        auto i_pos = i_ray(t);
        float sigma = accessor.getValue(nanovdb::Coord::Floor(i_pos));
        if (rnd(seed) > sigma / max_value) {
          continue;
        } else {
          break;
        }
      }

      if (t >= i_ray.t1()) {
        miss = true;
        break;
      }

      // hit happens, sample a new direction
      auto next_origin = grid->indexToWorldF(i_ray(t));
      float3 next_dir;
      uniform_sample_sphere(rnd(seed), rnd(seed), next_dir);
      w_ray = nanovdb::Ray<float>(next_origin, float3_as_vec(next_dir));
      factor *= color;
    }

    if (miss) {
      contrib += factor * get_light_emission(scene, vec_as_float3(w_ray.dir()));
    }
  }
  image[index] = contrib / float(spp);
}

template <typename Kernel, typename... Args>
void launch2d(Kernel &&k, int width, int height, Args &&... args) {
  dim3 block_size(1, 1);
  int grid_x = (width + block_size.x - 1) / block_size.x;
  int grid_y = (height + block_size.y - 1) / block_size.y;
  dim3 grid_size(grid_x, grid_y);
  printf("width %d, height %d\n", width, height);
  k<<<grid_size, block_size>>>(args...);
}

#define CHECK_CUDA_ERROR                                                       \
  do {                                                                         \
    hipError_t e;                                                             \
    e = hipGetLastError();                                                    \
    if (e != hipSuccess) {                                                    \
      printf("CUDA ERROR\n");                                                  \
    }                                                                          \
  } while (false)

void render(const Scene &scene, float *d_image) {
  printf("render\n");
  assert(scene.volume_grid);
  // auto kernel = render_kernel_raymarching;
  auto kernel = render_kernel_delta_tracking;
  launch2d(
      kernel, scene.frame_width, scene.frame_height, scene, (float3 *)d_image);
}
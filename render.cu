#include "hip/hip_runtime.h"
#include "random.h"
#include "render.h"
#include "vec_math.h"
#include <nanovdb/util/Ray.h>
#include <stdio.h>

__device__ __forceinline__ float3 array_as_float3(const float *vec) {
  return make_float3(vec[0], vec[1], vec[2]);
}

__device__ __forceinline__ float3
vec_as_float3(const nanovdb::Vec3<float> &vec) {
  return make_float3(vec[0], vec[1], vec[2]);
}

__device__ __forceinline__ nanovdb::Vec3<float>
float3_as_vec(const float3 &vec) {
  return nanovdb::Vec3<float>(vec.x, vec.y, vec.z);
}

__device__ float ray_march_transmittance(const nanovdb::FloatGrid *grid,
                                         const nanovdb::Ray<float> &wRay,
                                         float dt) {
  // transform the ray to the grid's index-space...
  nanovdb::Ray<float> iRay = wRay.worldToIndexF(*grid);
  // clip to bounds.
  if (iRay.clip(grid->tree().bbox()) == false) {
    return 1.0f;
  }
  // get an accessor.
  auto acc = grid->tree().getAccessor();
  // integrate along ray interval...
  float transmittance = 1.0f;
  for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
    float sigma = acc.getValue(nanovdb::Coord::Floor(iRay(t)));
    transmittance *= 1.0f - sigma * dt;
  }
  return transmittance;
}

__device__ __forceinline__ uint32_t reverse_bits_32(uint32_t n) {
  n = (n << 16) | (n >> 16);
  n = ((n & 0x00ff00ff) << 8) | ((n & 0xff00ff00) >> 8);
  n = ((n & 0x0f0f0f0f) << 4) | ((n & 0xf0f0f0f0) >> 4);
  n = ((n & 0x33333333) << 2) | ((n & 0xcccccccc) >> 2);
  n = ((n & 0x55555555) << 1) | ((n & 0xaaaaaaaa) >> 1);
  return n;
}

__device__ __forceinline__ float radical_inverse_base_2(uint32_t n) {
  return saturate(reverse_bits_32(n) * float(2.3283064365386963e-10));
}

// 0 <= i < n
__device__ __forceinline__ float2 hammersley_sample(uint32_t i, uint32_t n) {
  return make_float2((float)(i + 1) / (float)(n), radical_inverse_base_2(i));
}

__device__ __forceinline__ float uniform_sample_cone_pdf(float cos_phi) {
  return 0.5f * InvPi / (1.0f - cos_phi);
}

__device__ __forceinline__ float
uniform_sample_cone(float u, float v, float cos_phi, float3 &d) {
  float theta = 2.0f * Pi * u;
  float y = 1.0f - v * (1.0f - cos_phi);
  auto r = sqrtf(max(0.0f, 1.0f - y * y));
  d.x = r * cosf(theta);
  d.y = y;
  d.z = r * sinf(theta);
  return uniform_sample_cone_pdf(cos_phi);
}

// y goes upward
__device__ __forceinline__ float
uniform_sample_hemisphere(float u, float v, float3 &d) {
  return uniform_sample_cone(u, v, 0, d);
}

__device__ __forceinline__ float
uniform_sample_hemisphere(unsigned int &randState, float3 &d) {
  return uniform_sample_hemisphere(rnd(randState), rnd(randState), d);
}

__device__ __forceinline__ float
uniform_sample_sphere(float u, float v, float3 &d) {
  return uniform_sample_cone(u, v, -1.0f, d);
}

// y goes upward
__device__ __forceinline__ float
cosine_sample_hemisphere(float u, float v, float3 &d) {
  auto theta = 2.0f * Pi * u;
  auto r = sqrtf(v);
  auto y = sqrtf(max(1 - r * r, 0.0f));
  d.x = r * cosf(theta);
  d.y = y;
  d.z = r * sinf(theta);
  return y * InvPi;
}

__device__ __forceinline__ float
cosine_sample_hemisphere(unsigned int &randState, float3 &d) {
  return cosine_sample_hemisphere(rnd(randState), rnd(randState), d);
}

__device__ __forceinline__ float lerp(float a, float b, float t) {
  return a + t * (b - a);
}

__device__ __forceinline__ float4 lerp(float4 a, float4 b, float4 t) {
  return make_float4(lerp(a.x, b.x, t.x),
                     lerp(a.y, b.y, t.y),
                     lerp(a.z, b.z, t.z),
                     lerp(a.w, b.w, t.w));
}

__device__ __forceinline__ float4 lerp(float a, float b, float4 t) {
  return lerp(make_float4(a, a, a, a), make_float4(b, b, b, b), t);
}

__device__ __forceinline__ float inverse_lerp(float a, float b, float v) {
  return (v - a) / (b - a);
}

__device__ __forceinline__ float clamp(float v, float a, float b) {
  return max(min(v, b), a);
}

__device__ __forceinline__ float
sample_array(float *data, uint32_t count, float u) {
  uint32_t i = clamp(count * u, 0, count - 2);
  return lerp(data[i], data[i + 1], saturate(count * u - i));
}

__device__ __forceinline__ nanovdb::Ray<float> sample_camera_ray(
    const Scene &scene, int c, int r, float2 jitter = make_float2(0.0f, 0.0f)) {
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;

  float2 uv = make_float2((float(c) + jitter.x) / float(frame_width),
                          (float(r) + jitter.y) / float(frame_height));
  float aspect = float(frame_width) / float(frame_height);

  auto &camera_pos = scene.camera_pos;
  nanovdb::Vec3<float> origin(camera_pos[0], camera_pos[1], camera_pos[2]);
  nanovdb::Vec3<float> direction(uv.x - 0.5f, (uv.y - 0.5f) / aspect, -1.0);
  direction.normalize();
  nanovdb::Ray<float> wRay(origin, direction);
  return wRay;
}

__global__ void render_kernel_raymarching(Scene scene, float3 *image) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int r = blockIdx.y * blockDim.y + threadIdx.y;
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;
  const int index = r * frame_width + c;

  if ((c >= frame_width) || (r >= frame_height))
    return;

  nanovdb::Vec3<float> light_direction(
      scene.light_dir[0], scene.light_dir[1], scene.light_dir[2]);
  light_direction.normalize();
  auto grid = scene.volume_grid;

  nanovdb::Ray<float> wRay = sample_camera_ray(scene, c, r);
  // transform the ray to the grid's index-space...
  nanovdb::Ray<float> iRay = wRay.worldToIndexF(*grid);
  // clip to bounds.
  if (iRay.clip(grid->tree().bbox()) == false) {
    image[index] = make_float3(0, 0, 0);
    return;
  }
  // get an accessor.
  auto acc = grid->tree().getAccessor();
  // integrate along ray interval...
  float transmittance = 1.0f;
  float3 contrib = make_float3(0.0f, 0.0f, 0.0f);
  float dt = 0.5f;
  float3 color = make_float3(scene.phase_func.color[0],
                             scene.phase_func.color[1],
                             scene.phase_func.color[2]);

  for (float t = iRay.t0(); t < iRay.t1(); t += dt) {
    auto iPos = iRay(t);
    float sigma = acc.getValue(nanovdb::Coord::Floor(iPos));
    contrib += ray_march_transmittance(
                   grid, {grid->indexToWorldF(iPos), light_direction}, dt) *
               transmittance * dt * sigma * color * 0.25f * InvPi;
    transmittance *= 1.0f - sigma * dt;
  }
  image[index] = contrib;
}

struct TBN {
  float3 n;
  float3 t;
  float3 b;

  __device__ __forceinline__ float3 local_to_world(const float3 &v) const {
    return n * v.y + t * v.x + b * v.z;
  }
};

__device__ __forceinline__ TBN construct_tbn(const float3 &normal) {
  float3 n = normalize(normal);
  float3 up = make_float3(0.0f, 0.0f, 1.0f);
  if (n.z > 0.98f) {
    up = make_float3(1.0f, 0.0f, 0.0f);
  }
  float3 t = normalize(cross(up, n));
  float3 b = normalize(cross(t, n));
  TBN tbn{};
  tbn.n = n;
  tbn.t = t;
  tbn.b = b;
  return tbn;
}

__device__ __forceinline__ float3 get_light_emission(const Scene &scene,
                                                     const float3 &direction) {
  float3 light_dir = normalize(array_as_float3(scene.light_dir));
  float attenuation =
      dot(light_dir, direction) >= scene.light_cos_angle ? 1.0f : 0.0f;
  return attenuation * array_as_float3(scene.light_color);
}

__device__ __forceinline__ float sample_light(const Scene &scene,
                                              unsigned int &seed,
                                              float3 &direction,
                                              float3 &emission) {
  float3 local_dir;
  float pdf = uniform_sample_cone(
      rnd(seed), rnd(seed), scene.light_cos_angle, local_dir);
  float3 light_dir = normalize(array_as_float3(scene.light_dir));

  direction = construct_tbn(light_dir).local_to_world(local_dir);
  emission = array_as_float3(scene.light_color);
  return pdf;
}

__device__ __forceinline__ float sample_light_pdf(const Scene &scene,
                                                  float3 &direction) {
  float3 light_dir = normalize(array_as_float3(scene.light_dir));
  float attenuation =
      dot(light_dir, direction) >= scene.light_cos_angle ? 1.0f : 0.0f;
  return uniform_sample_cone_pdf(scene.light_cos_angle) * attenuation;
}

__device__ __forceinline__ float sample_free_flight(float u, float mu) {
  return -logf(1.0f - u) / mu;
}

__device__ __forceinline__ float eval_phase_function(float diffuse,
                                                     float cos_angle,
                                                     const float3 &curr_dir,
                                                     const float3 &next_dir) {
  float uniform_pdf = 0.25f * InvPi;
  float cone_pdf = 0.0f;
  if (dot(curr_dir, next_dir) > cos_angle) {
    cone_pdf = uniform_sample_cone_pdf(cos_angle);
  }

  return lerp(cone_pdf, uniform_pdf, clamp(diffuse, 0.0f, 1.0f));
}

__device__ __forceinline__ float sample_phase_function(unsigned int &seed,
                                                       float diffuse,
                                                       float cos_angle,
                                                       const float3 &curr_dir,
                                                       float3 &next_dir) {
  if (rnd(seed) < diffuse) {
    uniform_sample_sphere(rnd(seed), rnd(seed), next_dir);
  } else {
    float3 next_dir_local;
    uniform_sample_cone(rnd(seed), rnd(seed), cos_angle, next_dir_local);
    next_dir = construct_tbn(curr_dir).local_to_world(next_dir_local);
  }
  return eval_phase_function(diffuse, cos_angle, curr_dir, next_dir);
}

struct DeltaTrackResult {
  bool miss = false;
  float t = 0;
  float null_scatter_factor = 1.0f;
};

template <bool force_null_scatter = false>
__device__ __forceinline__ DeltaTrackResult
delta_track_ray(const nanovdb::FloatGrid *grid,
                const decltype(grid->getAccessor()) &accessor,
                unsigned int &seed,
                float max_sigma,
                float sigma_scale,
                nanovdb::Ray<float> &i_ray) {
  DeltaTrackResult res{};
  res.t = i_ray.t0();
  if (i_ray.clip(grid->tree().bbox()) == false) {
    res.miss = true;
    return res;
  }

  float &t = res.t;
  float &factor = res.null_scatter_factor;
  float last_null_scatter_factor = 1.0f;

  while (t < i_ray.t1()) {
    float scaled_max_value = max(max_sigma * sigma_scale, 1e-4);
    t += sample_free_flight(rnd(seed), scaled_max_value);
    factor *= last_null_scatter_factor;

    auto i_pos = i_ray(t);
    float sigma = accessor.getValue(nanovdb::Coord::Floor(i_pos)) * sigma_scale;
    float real_scatter_p = sigma / scaled_max_value;
    last_null_scatter_factor = 1.0f - real_scatter_p;

    if constexpr (force_null_scatter) {
      continue;
    }
    if (rnd(seed) > real_scatter_p) {
      continue;
    } else {
      break;
    }
  }

  if (t >= i_ray.t1()) {
    res.miss = true;
  }

  return res;
}

template <RenderMode mode>
__global__ void render_kernel_delta_tracking(Scene scene, float3 *image) {
  const int c = blockIdx.x * blockDim.x + threadIdx.x;
  const int r = blockIdx.y * blockDim.y + threadIdx.y;
  int frame_width = scene.frame_width;
  int frame_height = scene.frame_height;
  const int index = r * frame_width + c;

  if ((c >= frame_width) || (r >= frame_height))
    return;

  unsigned int seed = tea<4>(index, 11424);
  int spp = scene.spp;
  float aspect = float(frame_width) / float(frame_height);
  auto grid = scene.volume_grid;
  float max_value = scene.max_value;
  float sigma_scale = 1.0f;
  float3 color = make_float3(scene.phase_func.color[0],
                             scene.phase_func.color[1],
                             scene.phase_func.color[2]);
  float3 contrib = make_float3(0.0f, 0.0f, 0.0f);
  auto accessor = grid->tree().getAccessor();
  for (int i = 0; i < spp; i++) {
    float2 jitter = hammersley_sample(i, spp);
    nanovdb::Ray<float> w_ray = sample_camera_ray(scene, c, r, jitter);
    float3 factor = make_float3(1.0f, 1.0f, 1.0f);

    DeltaTrackResult hit{};
    float last_phase = 1.0f;
    int bounce = 0;
    for (; !hit.miss && bounce < 30; bounce++) {
      nanovdb::Ray<float> i_ray = w_ray.worldToIndexF(*grid);
      hit = delta_track_ray<false>(
          grid, accessor, seed, max_value, sigma_scale, i_ray);
      if (hit.miss) {
        break;
      }
      // hit happens, sample a new direction
      auto next_origin = grid->indexToWorldF(i_ray(hit.t));
      float3 curr_dir = normalize(vec_as_float3(w_ray.dir()));
      float3 next_dir;
      last_phase = sample_phase_function(seed,
                                         scene.phase_func.diffuse,
                                         scene.phase_func.cos_angle,
                                         curr_dir,
                                         next_dir);
      w_ray = nanovdb::Ray<float>(next_origin, float3_as_vec(next_dir));
      factor *= color;

      if constexpr (mode == RenderMode::RatioTracking ||
                    mode == RenderMode::MIS) {
        float3 light_dir;
        float3 light_emission;
        float pe = sample_light(scene, seed, light_dir, light_emission);
        nanovdb::Ray<float> w_light_ray(next_origin,
                                        float3_as_vec(normalize(light_dir)));
        nanovdb::Ray<float> i_light_ray = w_light_ray.worldToIndexF(*grid);
        auto rt = delta_track_ray<true>(
            grid, accessor, seed, max_value, sigma_scale, i_light_ray);
        float phase = eval_phase_function(scene.phase_func.diffuse,
                                          scene.phase_func.cos_angle,
                                          curr_dir,
                                          normalize(light_dir));
        float3 rt_contrib =
            factor * phase * rt.null_scatter_factor * light_emission / pe;
        if constexpr (mode == RenderMode::RatioTracking) {
          contrib += rt_contrib;
        }

        if constexpr (mode == RenderMode::MIS) {
          // balance heuristic
          float w_rt = pe / (pe + rt.null_scatter_factor * phase);
          contrib += rt_contrib * w_rt;
        }
      }
    }

    if (hit.miss) {
      if (mode != RenderMode::RatioTracking || bounce == 0) {
        // pure ratio tracking only handles one or more bounces
        float w_dt = 1.0f;
        if (mode == RenderMode::MIS && bounce != 0) {
          float pe =
              sample_light_pdf(scene, normalize(vec_as_float3(w_ray.dir())));
          w_dt = hit.null_scatter_factor * last_phase /
                 (pe + hit.null_scatter_factor * last_phase);
        }
        contrib += w_dt * factor *
                   get_light_emission(scene, vec_as_float3(w_ray.dir()));
      }
    }
  }
  image[index] = contrib / float(spp);
}

template <typename Kernel, typename... Args>
void launch2d(Kernel &&k, int width, int height, Args &&... args) {
  dim3 block_size(1, 1);
  int grid_x = (width + block_size.x - 1) / block_size.x;
  int grid_y = (height + block_size.y - 1) / block_size.y;
  dim3 grid_size(grid_x, grid_y);
  printf("width %d, height %d\n", width, height);
  k<<<grid_size, block_size>>>(args...);
}

#define CHECK_CUDA_ERROR                                                       \
  do {                                                                         \
    hipError_t e;                                                             \
    e = hipGetLastError();                                                    \
    if (e != hipSuccess) {                                                    \
      printf("CUDA ERROR\n");                                                  \
    }                                                                          \
  } while (false)

void render(const Scene &scene, float *d_image) {
  printf("render\n");
  assert(scene.volume_grid);

  auto launch = [&](auto kernel) {
    launch2d(kernel,
             scene.frame_width,
             scene.frame_height,
             scene,
             (float3 *)d_image);
  };

  switch (scene.mode) {
  case RenderMode::DeltaTracking:
    launch(render_kernel_delta_tracking<RenderMode::DeltaTracking>);
    break;
  case RenderMode::RatioTracking:
    launch(render_kernel_delta_tracking<RenderMode::RatioTracking>);
    break;
  case RenderMode::MIS:
    launch(render_kernel_delta_tracking<RenderMode::MIS>);
    break;
  case RenderMode::SpectralMIS:
    launch(render_kernel_delta_tracking<RenderMode::SpectralMIS>);
    break;
  }

  hipDeviceSynchronize();
}